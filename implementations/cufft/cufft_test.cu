#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define NX 256
#define BATCH 1

void print_sequence(const hipfftComplex* arr)
{
    printf("[");
    for(size_t i = 0; i < NX; ++i)
    {
        printf((i == NX-1)
            ? "(%f + %fi)]\n"
            : "(%f + %fi), ", arr[i].x, arr[i].y);
    }
}

int main()
{
    printf("Hello World!\n");
    
    size_t data_size = sizeof(hipfftComplex)*NX*BATCH;
    hipfftComplex* data = (hipfftComplex*) malloc(data_size);
    hipfftComplex* gpu_data;
    hipError_t err;
    // Allocate GPU buffer
    if((err = hipMalloc((void**) &gpu_data, data_size)) != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Failed to allocate '%d'\n", err);
        return 1;
    }

    // Initializing input sequence
    for(size_t i = 0; i < NX; ++i)
    {
        data[i].x = i;
        data[i].y = 0.;
    }

    print_sequence(data);

    printf("%p %p %zu %d\n", gpu_data, data, sizeof(hipfftComplex), hipMemcpyHostToDevice);

    // Allocate GPU buffer
    if((err = hipMemcpy(gpu_data, data, data_size, hipMemcpyHostToDevice)) != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Failed to copy buffer to GPU '%d'\n", err);
        return 1;
    }

    hipfftHandle plan;
    hipfftResult_t res;
    if((res = hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH)) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT error: Plan creation failed '%d'\n", res);
        return 1;
    }
    
    /* Note:
    * Identical pointers to input and output arrays implies in-place
    transformation
    */
    if((res = hipfftExecC2C(plan, gpu_data, gpu_data, HIPFFT_FORWARD)) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT error: ExecC2C Forward failed '%d'\n", err);
        return 1;
    }

    if ((err = hipDeviceSynchronize()) != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Failed to synchronize\n");
        return;	
    }

    // // Print Computed FFT
    // printf("FFT:\n");
    // for(size_t i = 0; i < NX; ++i)
    // {
    //     printf("%f\n", data[i].x);
    // }
    
    if((res = hipfftExecC2C(plan, gpu_data, gpu_data, HIPFFT_BACKWARD)) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT error: ExecC2C Inverse failed '%d'\n", err);
        return 1;
    }

    // Retrieve computed FFT buffer
    if((err = hipMemcpy(data, gpu_data, data_size, hipMemcpyHostToDevice)) != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Failed to copy buffer to GPU '%d'\n", err);
        return 1;
    }

    // // Print Computed IFFT
    // printf("FFT:\n");
    // for(size_t i = 0; i < NX; ++i)
    // {
    //     printf("%f\n", data[i].x);
    // }

    hipfftDestroy(plan);
    hipFree(data);
}