#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <iostream>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define NX 8
#define BATCH 1

class Foo;

// #define CU_ERR_CHECK_MSG(err, msg) {               \
//             if(err != cudaSuccess) {               \
//                 fprintf(stderr, msg);              \
//                 exit(1);                           \
//             }                                      \
//         }

// #define CU_CHECK_MSG(res, msg) {                   \
//             if(res != CUFFT_SUCCESS) {             \
//                 fprintf(stderr, msg);              \
//                 exit(1);                           \
//             }                                      \
//         }

#define CU_ERR_CHECK_MSG(err, msg, ...) {          \
            if(err != hipSuccess) {               \
                fprintf(stderr, msg __VA_OPT__(,)  \
                    __VA_ARGS__);                  \
                exit(1);                           \
            }                                      \
        }

#define CU_CHECK_MSG(res, msg, ...) {              \
            if(res != HIPFFT_SUCCESS) {             \
                fprintf(stderr, msg __VA_OPT__(,)  \
                    __VA_ARGS__);                  \
                exit(1);                           \
            }                                      \
        }


void print_sequence(const hipfftComplex* arr)
{
    printf("[");
    for(size_t i = 0; i < NX; ++i)
    {
        printf((i == NX-1)
            ? "(%f + %fi)]\n"
            : "(%f + %fi), ", arr[i].x, arr[i].y);
    }
}

int main()
{
    
    size_t data_size = sizeof(hipfftComplex)*NX*BATCH;
    hipfftComplex* data = (hipfftComplex*) malloc(data_size);
    hipfftComplex* gpu_data;
    hipError_t err;
    
    // Allocate GPU buffer
    err = hipMalloc(&gpu_data, data_size);
    CU_ERR_CHECK_MSG(err, "Cuda error: Failed to allocate '%d'\n", err);

    // Initializing input sequence
    for(size_t i = 0; i < NX; ++i)
    {
        data[i].x = i;
        data[i].y = 0.;
    }

    print_sequence(data);

    // Copy data to GPU buffer
    err = hipMemcpy(gpu_data, data, data_size, hipMemcpyHostToDevice);
    CU_ERR_CHECK_MSG(err, "Cuda error: Failed to copy buffer to GPU '%d'\n", err);
    // CU_ERR_CHECK_MSG(err, "Cuda error: Failed to copy buffer to GPU\n");

    hipfftHandle plan;
    hipfftResult_t res;
    res = hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH);
    CU_CHECK_MSG(res, "CUFFT error: Plan creation failed '%d'\n", res);
    // CU_CHECK_MSG(res, "CUFFT error: Plan creation failed\n");
    
    /* Note:
    * Identical pointers to input and output arrays implies in-place
    transformation
    */

    // Execute Forward 1D FFT
    res = hipfftExecC2C(plan, gpu_data, gpu_data, HIPFFT_FORWARD);
    CU_CHECK_MSG(res, "CUFFT error: ExecC2C Forward failed '%d'\n", res);
    // CU_CHECK_MSG(res, "CUFFT error: ExecC2C Forward failed\n");

    // Await end of execution
    err = hipDeviceSynchronize();
    CU_ERR_CHECK_MSG(err, "Cuda error: Failed to synchronize\n");

    // Execute Inverse 1D FFT
    res = hipfftExecC2C(plan, gpu_data, gpu_data, HIPFFT_BACKWARD);
    CU_CHECK_MSG(res, "CUFFT error: ExecC2C Inverse failed '%d'\n", res);
    // CU_CHECK_MSG(res, "CUFFT error: ExecC2C Inverse failed\n");

    // Await end of execution
    err = hipDeviceSynchronize();
    CU_ERR_CHECK_MSG(err, "Cuda error: Failed to synchronize\n");

    // Retrieve computed FFT buffer
    err = hipMemcpy(data, gpu_data, data_size, hipMemcpyDeviceToHost);
    CU_ERR_CHECK_MSG(err, "Cuda error: Failed to copy buffer to GPU '%d'\n", err);

    // Divide result by N
    for(size_t i = 0; i < NX; ++i) {
        data[i].x /= NX;
    }


    hipfftDestroy(plan);
    hipFree(gpu_data);

    // Print Computed IFFT
    print_sequence(data);
}